#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#define M 20

// RED = 0, BLACK = 1
enum nodeColor {
    RED,
    BLACK
};

enum result {
    Success,
    Failure,
    FirstInsert
};

enum caseFlag {
    NOOP,
    DID_CASE1,
    DID_CASE3
};

struct par_rbNode {
    int key, color;
    int flag;
    struct par_rbNode *left, *right, *parent;
};

// /*Function prototypes */
__device__ void createNIL();
__device__ struct par_rbNode * createNode(int);
__device__ void createTree();
__device__ struct par_rbNode * Traverse(int);
__device__ enum result PlaceNode(struct par_rbNode *, struct par_rbNode *);
__device__ void Insert_Rebalance(struct par_rbNode *);
__device__ bool Update_Rotation(struct par_rbNode *, enum caseFlag *);
__device__ bool Left_Rotate(struct par_rbNode *);
__device__ bool Right_Rotate(struct par_rbNode *);

__device__ struct par_rbNode *nodes;
__device__ struct par_rbNode *root;
__device__ struct par_rbNode *NIL;
__device__ struct par_rbNode *rtParent;
__device__ struct par_rbNode *rtSibling; // U might feel this is unncessary, but it will be used
__device__ int nodeIndex = 0;
__device__ int tmpIndex = 2;
__device__ struct par_rbNode *tmp[M];// need M tmps

__device__ int createFlag = false;

__device__ void createNIL(){
    NIL = &nodes[0];
    NIL->color = BLACK;
    NIL->key = -1;
    NIL->left = NIL->right = NIL->parent = NIL;
    printf("NIL created\n");
}

__device__ struct par_rbNode * createNode(int key){

    bool ok;
    do{
        ok = atomicCAS(&createFlag,false,true); //Capture the lock
    }while(!ok);
    atomicAdd(&nodeIndex,1);
    atomicAdd(&tmpIndex,1);
    nodes[nodeIndex].key = key;
    nodes[nodeIndex].flag = true;
    nodes[nodeIndex].left = nodes[nodeIndex].right = nodes[nodeIndex].parent = NIL;
    tmp[tmpIndex] = &nodes[nodeIndex];
    createFlag = false;
    // atomicCAS(&createFlag,true,false); //Release the lock
    printf("Created %d\n",key);
    return tmp[tmpIndex]; // Even if this thread pauses it will eventually return the correct pointer

}

__device__ void createTree(){
    rtParent = createNode(-1);
    rtSibling = createNode(-1);
    // NIL = createNode(-1);
    root = NIL;
    rtParent->parent = NIL;
    rtSibling->parent = rtParent;
    rtSibling->right = NIL;
    rtSibling->left    = NIL;
    rtParent->left = root;
    //rtParent->left = root; Why only left, y not right?
    //ANS: Since we check for left parent condition first 
    //(if u don't understand, try to insert a node to a tree with only one node)
    rtParent->right = rtSibling;
    rtParent->flag = false;
    rtSibling->flag = false;
    rtParent->color = BLACK;
    rtSibling->color = BLACK;
    // NIL->left = NIL;
    // NIL->right = NIL;
    NIL->parent = rtParent;
    NIL->flag = false;
    // NIL->color = BLACK;
    printf("Tree Created \n");
}

__device__ struct par_rbNode * Traverse(int key){
    struct par_rbNode *x;
    struct par_rbNode *insertPoint = NIL;
    // struct par_rbNode *savert;
    bool success;

    // do{
    //  savert = root;
    //  success = DCAS(&root->flag,false,true,&root,savert,savert); //Catching the flag of the root
    // }while(!success);

    //An alternate for DCAS - should check if it works or not
    // do{
    //     savert = root;
    //     success = atomicCAS(&root->flag,false,true); //Catching the flag of the root
    // }while(savert!=root || !success);


    do{
        // savert = root;
        success = atomicCAS(&root->flag,false,true); //Catching the flag of the root
    }while(!success);
    //success => captured the root flag
    //savert != root => root has changed
    //!success => root is under lock
    //thread will come out of the loop only after "success" and "savert==root" 
    x = root;
    while(x != NIL){
        if(key == x->key) {
            x->flag = false; // Release the flag that was just caught
            return NULL; // Traversing is done. Node is already there so Insert() fails.
        }
        insertPoint = x;
        if(key < x->key){
            if(x->left != NIL){
                success = atomicCAS(&x->left->flag,false,true);
                if(!success){
                    x->flag = false; // Release the flag of x
                    return NULL;
                }//end if
                x = x->left;
                insertPoint->flag = false;  
            }else{
                x = x->left;
            }//end if
        }else{
            if(x->right != NIL){
                success = atomicCAS(&x->right->flag,false,true);
                if(!success){
                    x->flag = false;
                    return NULL;
                }//end if
                x = x->right;
                insertPoint->flag = false;
            }else{
                x = x->right;
            }//end if
        }//end if
    }//end while
    return insertPoint;
}

__device__ enum result PlaceNode(struct par_rbNode *newNode, struct par_rbNode *insertPoint){
    //flags on newNode and insertPoint are held
    bool ok;
    // struct par_rbNode *uncle,*savep;

    if(insertPoint == NIL){ //tree is empty
        newNode->color = BLACK;
        newNode->parent = rtParent;
        rtParent->left = newNode;
        root=newNode;
        NIL->flag = false; // release NIL node, that u caught during Traverse
        newNode->flag = false;
        return FirstInsert;
    }else{ // the tree is not empty so...
        newNode->parent = insertPoint;
        //set the flags of the grandparent and uncle
        if(insertPoint == insertPoint->parent->left){ //uncle is right child
            // savep = insertPoint->parent; // save parent ptr
            // uncle = savep->right;   // rtSibling is used here, when insertPoint is root
            ok = atomicCAS(&insertPoint->parent->flag,false,true);
            if(ok){
                ok = atomicCAS(&insertPoint->parent->right->flag,false,true);
                // if(ok){
                //     ok = atomicCAS(&insertPoint->parent,savep,savep) && atomicCAS(&savep->right,uncle,uncle);
                // }
                if(!ok){ //back off
                    insertPoint->parent->flag = false;
                    insertPoint->parent->right->flag = false;
                }else{
                    insertPoint->parent->flag = false;
                }//end if
            }
        }else{// uncle is left child
            // savep = insertPoint->parent; // save parent ptr
            // uncle = savep->left;
            ok = atomicCAS(&insertPoint->parent->flag,false,true);
            if(ok){
                ok = atomicCAS(&insertPoint->parent->left->flag,false,true);
                // if(ok){
                //     ok = atomicCAS(&insertPoint->parent,savep,savep) && atomicCAS(&savep->left,uncle,uncle);
                // }
                if(!ok){ //back off
                    insertPoint->parent->flag = false;
                    insertPoint->parent->left->flag = false;
                }else{
                    insertPoint->parent->flag = false;
                }//end if
            }
        }//end if
        if(!ok){
         // This "!ok" is when u fail to capture the grandparent flag,
         // u haven't caught any extra flags so just get rid of the flag of insertPoint
            newNode->parent = NIL;
            insertPoint->flag = false; // release flag
            return Failure;         //avoid deadlock 
        }
        // When u have successfully captured all the required flags.
        // i.e. parent, grandparent, uncle
        if(newNode->key < insertPoint->key){
            //insert as left child
            insertPoint->left = newNode;
            return Success;
        }else{//insertas right child
            insertPoint->right = newNode;
            return Success;
        }
    }
}

__device__ void Insert_Rebalance(struct par_rbNode *x){ //THIS FUNCTION DOESN'T BACKOFF. IT KEEPS TRYING
    //we hold flags on x, p(x), p(p(x)), and uncle(x)
    struct par_rbNode *oldx;
    struct par_rbNode *uncle, *olduncle;
    // struct par_rbNode *savep, *savegp;
    struct par_rbNode *brother;
    struct par_rbNode *nephew;
    bool ok;
    bool updateSucceeds; //Update-Rotation successded?

    //caseF is short for caseFlag (avoiding confusion between global enum and local variable)
    enum caseFlag caseF = NOOP; // initially not doing any case
    //define uncle for first iteration
    if(x->parent == x->parent->parent->left){
        uncle = x->parent->parent->right;
    }else{ // uncle is the left child not right
        uncle = x->parent->parent->left;
    }
    while((x != root) && (x->parent->color == RED)){
        //do color-update and/or rotaion as required
        do{
            updateSucceeds = Update_Rotation(x,&caseF);
        }while(!updateSucceeds);

        //CASE 1: move to grandparent after color update
        if(caseF == DID_CASE1){
            oldx = x;   //save pointer to the old x
            olduncle = uncle; // save pointer to old uncle;
            x = x->parent->parent; // up to grandparent
            do{ //find new uncle of x and get flags
                if(x->parent == x->parent->parent->left){
                    // savep = x->parent;
                    // savegp = savep->parent;
                    // uncle = savegp->right;
                    ok = atomicCAS(&x->parent->flag,false,true);
                    if(ok){
                        ok = atomicCAS(&x->parent->parent->flag,false,true);
                        if(ok){
                            ok = atomicCAS(&x->parent->parent->right->flag,false,true);
                            if(!ok){
                                x->parent->flag = false;
                                x->parent->parent->flag = false;
                                x->parent->parent->right->flag = false;
                            }else{
                                x->parent->flag = false;
                                x->parent->parent->flag = false;
                            }
                        }else{
                            x->parent->flag = false;
                        }
                    }
                }else{
                    // savep = x->parent;
                    // savegp = savep->parent;
                    // uncle = savegp->left;
                    ok = atomicCAS(&x->parent->flag,false,true);
                    if(ok){
                        ok = atomicCAS(&x->parent->parent->flag,false,true);
                        if(ok){
                            ok = atomicCAS(&x->parent->parent->left->flag,false,true);
                            if(!ok){
                                x->parent->flag = false;
                                x->parent->parent->flag = false;
                                x->parent->parent->left->flag = false;
                            }else{
                                x->parent->flag = false;
                                x->parent->parent->flag = false;
                            }
                        }else{
                            x->parent->flag = false;
                        }
                    }
                }
            }while(!ok); //THIS FUNCTION DOESN'T BACKOFF. IT KEEPS TRYING
            //Release old flags for CASE 1
            oldx->parent->flag = false;
            olduncle->flag = false;
            oldx->flag = false;
        }
    //in CASE 3 loop will exit: parent will be BLACK
    }
    switch(caseF){
        case NOOP: //In the beginning of this function we had 
                    //x,p(x),p(p(x)),uncle(x) - release them
                    x->parent->parent->flag = false;
                    x->parent->flag = false;
                    uncle->flag = false;
                    x->flag = false;
                    break;
        case DID_CASE1: //Release the last set of flags acquired
                    x->parent->parent->flag = false;
                    x->parent->flag = false;
                    uncle->flag = false;
                    x->flag = false;
                    break;
        case DID_CASE3: //release flags on ROTATED x, etc
                    if(x == x->parent->left){
                        brother = x->parent->right;
                        nephew = x->parent->right->right;
                    }else{
                        brother = x->parent->left;
                        nephew = x->parent->left->left;
                    }
                    x->parent->flag = false;
                    brother->flag = false;
                    nephew->flag = false;
                    x->flag = false;
                    break;
    }
    root->color = BLACK; 
}

__device__ bool Update_Rotation(struct par_rbNode *x, enum caseFlag *caseF){
    //we hold flags on x, p(x), p(p(x)) and uncle(x)
    struct  par_rbNode *xUncle;
    struct  par_rbNode *oldx; //*ggp; // ggp -> greatgrandparent
    bool ok;

    if(x->parent == x->parent->parent->left){
        //the parent is a left child
        xUncle = x->parent->parent->right;
        if(xUncle->color == RED){
            //CASE 1 - recoloring
            // U have all the flags u need. So this is simple, similar to serial code
            x->parent->color = BLACK;
            xUncle->color = BLACK;
            x->parent->parent->color = RED;
            *caseF = DID_CASE1;
            return true; // This true is for "updateSucceeds"
        }else{ // rotation(s) will be needed
            if(x == x->parent->right){//CASE2
                oldx = x; // save old x in case rotate fails
                x = x->parent;
                ok = Left_Rotate(x);
                if(!ok){
                    x = oldx; //undo change to x
                    return false; //This false is for "updateSucceeds"
                }
            }
            //In CASE 3, if the right-rotation fails,
            //CASE 3 fails but the algorithm still works
            //beacuse the process will return false to 
            //Insert_Rebalance, and Insert_Rebalance will
            //call Update_Rotation again to complete CASE3
            do{ // get great grandparent's flag
                // ggp = x->parent->parent->parent;
                ok = atomicCAS(&x->parent->parent->parent->flag,false,true);
            }while(!ok);    //KEEPS TRYING, DOESN'T BACK OFF
            ok = Right_Rotate(x->parent->parent);
            if(!ok){
                x->parent->parent->parent->flag = false;
                return false; //This false is for "updateSucceeds"
            }else{
                x->parent->color = BLACK;
                x->parent->right->color = RED;
                *caseF = DID_CASE3;
                x->parent->parent->parent->flag = false; //remove the ggp flag as rotation was successful
                return true;
            }
        } 
        //symmetric to above code
    }else{
        //the parent is a right child
        xUncle = x->parent->parent->left;
        if(xUncle->color == RED){
            //CASE 1 - recoloring
            // U have all the flags u need. So this is simple, similar to serial code
            x->parent->color = BLACK;
            xUncle->color = BLACK;
            x->parent->parent->color = RED;
            *caseF = DID_CASE1;
            return true;
        }else{ // rotation(s) will be needed
            if(x == x->parent->left){//CASE2
                oldx = x; // save old x in case rotate fails
                x = x->parent;
                ok = Right_Rotate(x);
                if(!ok){
                    x = oldx; //undo change to x
                    return false;
                }
            }
            //In CASE 3, if the left-rotation fails,
            //CASE 3 fails but the algorithm still works
            //beacuse the process will return false to 
            //Insert_Rebalance, and Insert_Rebalance will
            //call Update_Rotation again to complete CASE3
            do{ // get great grandparent's flag
                // ggp = x->parent->parent->parent;
                ok = atomicCAS(&x->parent->parent->parent->flag,false,true);
            }while(!ok);
            ok = Left_Rotate(x->parent->parent);
            if(!ok){
                x->parent->parent->parent->flag = false;
                return false;
            }else{
                x->parent->color = BLACK;
                x->parent->left->color = RED;
                *caseF = DID_CASE3;
                x->parent->parent->parent->flag = false;
                return true;
            }
        }
    }
}

//A rotation will always be successful(true), as u can reach the rotate command
//only after u have cptured all the requried flags

__device__ bool Left_Rotate(struct par_rbNode *z){
    //z is the root of the rotation subtree. The locks
    // held at this point are : z,z->parent and z->right (and sibling of z but its not useful here)
    // bool ok;
    struct par_rbNode *zrl,*zr;

    if(z->parent == rtParent){
        //rotating at the root
        zrl = z->right->left;
        zr = z->right;
        // if a process has set the flag of a node q,
        //no other process can move one of the children of q away from q
        zrl->parent = z;
        z->right = zrl;
        // ok = CAS3(z->right,zrl,z->right,
        //      z->right,z,zrl->parent,
        //      zrl,zrl,z->right->left);
        //update other links
        root = zr;
        rtParent->left = root;
        root->parent = rtParent;
        z->parent = root;
        root->left = z;
    }else{
        //rotating under the root (parent, etc . exist)
        if(z == z->parent->left){
            //z is left child
            zrl = z->right->left;
            zr = z->right;
            // if a process has set the flag of a node q,
            //no other process can move one of the children of q away from q
            zrl->parent = z;
            z->right = zrl;
            //update other links
            z->parent->left = zr;
            z->right->parent = z->parent;
            z->parent = zr;
            z->right->left = z;
        }else{
            // z is right child
            zrl = z->right->left;
            zr = z->right;
            // if a process has set the flag of a node q,
            //no other process can move one of the children of q away from q
            zrl->parent = z;
            z->right = zrl;
            //update other links
            z->parent->right = zr;
            z->right->parent = z->parent;
            z->parent = zr;
            z->right->left = z;
        }
    }
    return true;
}

//symmetric to Left_rotate
__device__ bool Right_Rotate(struct par_rbNode *z){
    //z is the root of the rotation subtree. The locks
    // held at this point are : z,z->parent and z->left (and sibling of z but its not useful here)
    // bool ok;
    struct par_rbNode *zrl,*zr;

    if(z->parent == rtParent){
        //rotating at the root
        zrl = z->left->right;
        zr = z->left;
        // if a process has set the flag of a node q,
        //no other process can move one of the children of q away from q
        zrl->parent = z;
        z->left = zrl;
        // ok = CAS3(z->left,zrl,z->left,
        //      z->left,z,zrl->parent,
        //      zrl,zrl,z->left->right);
        //update other links
        root = zr;
        rtParent->right = root;
        root->parent = rtParent;
        z->parent = root;
        root->right = z;
    }else{
        //rotating under the root (parent, etc . exist)
        if(z == z->parent->right){
            //z is right child
            zrl = z->left->right;
            zr = z->left;
            // if a process has set the flag of a node q,
            //no other process can move one of the children of q away from q
            zrl->parent = z;
            z->left = zrl;
            //update other links
            z->parent->right = zr;
            z->left->parent = z->parent;
            z->parent = zr;
            z->left->right = z;
        }else{
            // z is left child
            zrl = z->left->right;
            zr = z->left;
            // if a process has set the flag of a node q,
            //no other process can move one of the children of q away from q
            zrl->parent = z;
            z->left = zrl;
            //update other links
            z->parent->left = zr;
            z->left->parent = z->parent;
            z->parent = zr;
            z->left->right = z;
        }
    }
    return true;
}

__device__ void Insert(int key){
    struct par_rbNode *newNode = createNode(key); //Internally the flag of the newNode is held
    struct par_rbNode *insertPoint;
    // enum result {Success,Failure,FirstInsert};
    // Create and initialize the new node
    enum result res;
    //insert the new node
    do{
        //Traverse tree to find insertion point
        insertPoint = Traverse(key);
        if(insertPoint != NULL){
            //add new node to tree
            res = PlaceNode(newNode,insertPoint);
            printf("%d\n",res);
            // res is short for result (avoiding confusion b/w global enum and local variable)
            if(res == Success){
                //node was added succcessfully so make 
                //tree red-black again by doing the 
                //necessary color updates and rotations
                Insert_Rebalance(newNode);
            }
        }else{
            break;
        }
    }while(!(res==Success)||(res==FirstInsert));
}

//Functions for printing the tree
__device__ void printPreorder(struct par_rbNode* node)
{
    if (node == NIL)
        return;
    /* first print the data of node */
    printf("%d-", node->key);
    printf("%d", node->color);
    printf("  ");  
    /* then recur on left child */
    printPreorder(node->left);
    /* now recur on right child */
    printPreorder(node->right);
}

__device__ void printInorder(struct par_rbNode* node)
{
    if (node == NIL)
        return;
    /* first recur on left child */
    printInorder(node->left);
    /* then print the data of node */
    printf("%d-", node->key);
    printf("%d", node->color);
    printf("  ");  
    /* now recur on right child */
    printInorder(node->right);
}

__device__ void printPostorder(struct par_rbNode* node)
{
    if (node == NIL)
        return;
    /* first recur on left child */
    printPostorder(node->left);
    /* then recur on right child */
    printPostorder(node->right);
    /* now print the data of node */
    printf("%d-", node->key);
    printf("%d", node->color);
    printf("  ");  
}

__device__ int threadsFinished = 0;
__device__ int passCreate = 0;

__global__ void RBT(struct par_rbNode *d_nodes) {

    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int threadCount = gridDim.x*blockDim.x;
    
    if(id == 0){
        printf("Starting the Tree\n");
        nodes = d_nodes; // Make it a global variable
        createNIL();
        createTree();
        atomicAdd(&passCreate,1);
    }

    while(1){
        if(passCreate){
            Insert(id);
            break;
        }
    }

    // //Print the time
    // //This will keep track of number of threads that are done
    atomicAdd(&threadsFinished,1);
    // // //Print the tree after all the threads are done
    if(threadsFinished == threadCount){
        if(id == 0){
            printf("PreOrder: ");
            printPreorder(root);
            printf("\n");
            printf("\n");
            printf("InOrder: ");
            printInorder(root);
            printf("\n");
            printf("\n");
            printf("PostOrder: ");
            printPostorder(root);
            printf("\n");
            printf("\n");
            printf("Done\n");
        }
    }
//return to main
}

int main() {
    struct par_rbNode h_nodes[M];
    struct par_rbNode *d_nodes;
    float time;
    // 1. Allocate device array.
    hipMalloc(&d_nodes, M * sizeof(struct par_rbNode));
    for(int i=0;i<M;i++){
            h_nodes[i].flag = false;
            h_nodes[i].color = RED;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 2. Copy array contents from host to device.
    hipMemcpy(d_nodes, h_nodes, M * sizeof(struct par_rbNode), hipMemcpyHostToDevice);
    printf("Kernel Launched\n");
    hipEventRecord(start, 0);
    RBT<<<1,1>>>(d_nodes);
    hipMemcpy(h_nodes, d_nodes, M * sizeof(struct par_rbNode), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    printf("Came back\n");
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for the kernel: %f ms\n", time);
    return 0;
}